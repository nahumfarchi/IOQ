
#include <hip/hip_runtime.h>
__global__ void add2( double * v1, const double * v2 ) 
{
    int idx = threadIdx.x;
    v1[idx] += v2[idx];
}

__global__ void add_mat(double * Lp, const double * dlp, const double * b, int rows, int cols, int numel)
{
    // Which block are we?
    //size_t const globalBlockIndex = blockIdx.x + blockIdx.y * gridDim.x;
    // Which thread are we within the block?
    //size_t const localThreadIdx = threadIdx.x + blockDim.x * threadIdx.y;
    // How big is each block?
    //size_t const threadsPerBlock = blockDim.x*blockDim.y;
    // Which thread are we overall?
    //size_t const globalThreadIdx = localThreadIdx + globalBlockIndex*threadsPerBlock;

    //if (globalThreadIdx >= numel) {
    //    return;
    //}

    //A[globalThreadIdx] = A[globalThreadIdx] + B[globalThreadIdx];
    size_t const i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= rows)
        return;
    size_t const j = blockDim.y * blockIdx.y + threadIdx.y;
    if (j >= cols)
        return;
    size_t const idx = i + j * rows;
    if (idx >= numel)
        return;
    Lp[idx] = dlp[i] + dlp[j] + b[i] - b[j] - 2*Lp[idx];
}

__global__ void lattice_inner_loop(
    float * out,
    float * Lp, 
    const float * dlp, 
    const float * b, 
    int rows, 
    int cols, 
    int numel)
{
    size_t const i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= rows)
        return;
    size_t const j = blockDim.y * blockIdx.y + threadIdx.y;
    if (j >= cols)
        return;
    size_t const idx = i + j * rows;
    if (idx >= numel)
        return;
    out[idx] = dlp[i] + dlp[j] + b[i] - b[j] - 2*Lp[idx];
}