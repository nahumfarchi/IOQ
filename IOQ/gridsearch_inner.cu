
#include <hip/hip_runtime.h>
#include <math.h>

#define SZ 32

__global__ 
void lattice_inner_loop(
    float * out,
    float * Lp, 
    const float * dlp, 
    const float * b, 
    const size_t rows, 
    const size_t cols, 
    const size_t numel)
{
    size_t const i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= rows)
        return;
    size_t const j = blockDim.y * blockIdx.y + threadIdx.y;
    if (j >= cols)
        return;
    size_t const idx = i + j * rows;
    if (idx >= numel)
        return;
    out[idx] = dlp[i] + dlp[j] + b[i] - b[j] - 2*Lp[idx];
}

__device__ float infinity(void)
{
    return 0x7f800000;
}

__device__ float minus_infinity(void)
{
    return 0xff800000;
}

/*
Find the minimum of
	b[r] - b[c] + R_rc
	r = 1...rows
	c = 1...cols
where R_rc is the resistance defined by (Lp)_rr + (Lp)_cc - 2 (Lp)_rc
*/
__global__ 
void reduce_cols(
    float * out_min,
    unsigned int * out_r,
    const float * Lp,
    const float * dlp,
    const float * b,
    unsigned int rows,
    unsigned int cols,
    unsigned int numel)
{
    unsigned int c = blockDim.y * blockIdx.y + threadIdx.y;
    if (c >= cols)
        return;
    unsigned int idx = c * rows;

    float min = infinity();
    unsigned int min_r = 0;

    for (unsigned int r = 0; r < rows; r++, idx++)
    {
        float z = dlp[r] + dlp[c] + b[r] - b[c] - 2*Lp[idx];
		//float z = dlp[r] + b[r] - 2*Lp[idx];
        if (z < min)
        {
            min = z;
            min_r = r;
        }
    }

    out_min[c] = min;
    out_r[c] = min_r;
}

/*
Find the minimum of
	b[r] - b[c] + R_rc
	r = 1...rows
	c = 1...cols
where R_rc is the resistance defined by (Lp)_rr + (Lp)_cc - 2 (Lp)_rc. Lp is the pseudo-inverse of the Laplacian.

R is a symmetric matrix given as an array of size nv*(nv-1)/2. This array was created by taking the columns of lower triangle part (not including the diagonal!)
For example:
	* * * *
	1 * * *
	2 5 * *
	3 6 8 *
will be given as 
    [1, 2, ..., 6]
*/
__global__ 
void reduce_cols_R_symmetric(
    float * out_min,
    unsigned int * out_r,
    const float * R,
    const float * b,
    unsigned int rows,
    unsigned int cols,
    unsigned int numel)
{
	float resistance;
    unsigned int c = blockDim.y * blockIdx.y + threadIdx.y;
    if (c >= cols)
        return;
    unsigned int idx;

    float min = infinity();
    unsigned int min_r = 0;

    for (unsigned int r = 0; r < rows; r++)
    {
		if (r < c) {
			idx = ((2*rows - r - 1)*r) / 2.0 + c - r - 1;
			resistance = R[idx];
		}
		else if (r > c) {
			idx = ((2*rows - c - 1)*c) / 2.0 + r - c - 1;
			resistance = R[idx];
		}
		else {
			resistance = 0;
		}
		
		float z = b[r] - b[c] + resistance;
        if (z < min)
        {
            min = z;
            min_r = r;
        }
    }

    out_min[c] = min;
    out_r[c] = min_r;
}

__global__ 
void reduce_cols_R(
    float * out_min,
    unsigned int * out_r,
    const float * R,
    const float * b,
    unsigned int rows,
    unsigned int cols,
    unsigned int numel)
{
    unsigned int c = blockDim.y * blockIdx.y + threadIdx.y;
    if (c >= cols)
        return;
    unsigned int idx = c * rows;;

    float min = infinity();
    unsigned int min_r = 0;

    for (unsigned int r = 0; r < rows; r++, idx++)
    {
        //float z = dlp[r] + dlp[c] + b[r] - b[c] - 2*Lp[idx];
		//float z = dlp[r] + b[r] - 2*Lp[idx];
		float z = b[r] - b[c] + R[idx];
        if (z < min)
        {
            min = z;
            min_r = r;
        }
    }

    out_min[c] = min;
    out_r[c] = min_r;
}

/* 
Find the minimum of 
	b_r - b_c + R_rc
	r = 1...rows
	c = 1...cols
where R_rc is the resistance matrix, approximated by 
	| Ztilde(:, r) - Ztilde(:, c) |^2
*/
__global__ 
void reduce_cols_Ztilde(
	float * out_min,
	unsigned int * out_r,
	const float * Ztilde,
	const unsigned int k,
	const float * b,
	const unsigned int rows,
	const unsigned int cols,
	const unsigned int numel)
{
	unsigned int c = blockDim.y * blockIdx.y + threadIdx.y;
    if (c >= cols)
        return;
    unsigned int idx = c * rows;
	float min = infinity();
    unsigned int min_r = 0;
	
	const unsigned int ck = c * k;
		
	for (unsigned int r = 0; r < rows; r++, idx++)
	{
		float resistance = 0;
		unsigned int zidx1 = ck;
		unsigned int zidx2 = r * k;
		//for (unsigned int i = 0; i < k; i++, zidx1++, zidx2++)
		for(; zidx1 < ck + k; zidx1++, zidx2++)
		{
			//resistance += ( Ztilde[zidx1] - Ztilde[zidx2] ) * ( Ztilde[zidx1] - Ztilde[zidx2] );
			resistance += pow( Ztilde[zidx1] - Ztilde[zidx2], 2 );
		}
			
		float z = b[r] - b[c] + resistance;
		//float z = b[r] + resistance;
		if (z < min)
		{
			min = z;
			min_r = r;
		}
	}
	
	out_min[c] = min;
    out_r[c] = min_r;
}

/* 
*Row major* pairwise Euclidean distances (squared)
input matrix in of size n x m
output matrix out of size n x n
*/
__global__ void matrix_euclidean_distance_kernel_fast(float* out, float* in, int n, int m){
	__shared__ float Ys[SZ][SZ];
	__shared__ float Xs[SZ][SZ];

	int bx = blockIdx.x, by = blockIdx.y;
	int tx = threadIdx.x, ty = threadIdx.y;

	int yBegin = by * SZ * m;
	int xBegin = bx * SZ * m;

	int yEnd = yBegin + m - 1, y, x, k, o;

	float tmp, s = 0;

	for (y = yBegin, x = xBegin;
		y <= yEnd;
		y += SZ, x += SZ){
		Ys[ty][tx] = in[y + ty * m + tx];
		Xs[tx][ty] = in[x + ty * m + tx];
		__syncthreads();

		for (k = 0; k<SZ; k++){
			tmp = Ys[ty][k] - Xs[k][tx];
			s += tmp * tmp;
		}
		__syncthreads();
	}
	o = by * SZ * n + ty * n + bx * SZ + tx;
	out[o] = s;
}

__global__ void
gpuPdist1(float *out, float *in, int n, int m)
{
	extern __shared__ float Rs[];
	float tmp, s;
	int myRow = blockIdx.x*256 + threadIdx.x;
	
	for(int r=0; r<n; r++) {
		s = 0;
		for(int i=0; i<=m/256; i++) {
			if (i*256+threadIdx.x < m)
				Rs[i*256+threadIdx.x] = in[r*m+i*256+threadIdx.x];
		}
		__syncthreads();
		
		for(int i=0; i<m && myRow<n; i++) {
			tmp = Rs[i] - in[myRow*m + i];
			s += tmp*tmp;
		}
		if (myRow < n)
			out[myRow*n+r] = s; // not sqrtf(s)
		__syncthreads();
	}
}

__global__ void
gpuPdist2(float *out, float *in, int n, int m) {
	__shared__ float Ys[SZ][SZ];
	__shared__ float Xs[SZ][SZ];
	int bx = blockIdx.x, by = blockIdx.y;
	int tx = threadIdx.x, ty = threadIdx.y;
	int yBegin = by*SZ*m;
	int xBegin = bx*SZ*m;
	int yEnd = yBegin + m - 1, y, x, k, o;
	float tmp, s = 0;
	
	for(y=yBegin, x=xBegin; y<=yEnd; y+=SZ, x+=SZ) {
		Ys[ty][tx] = in[y + ty*m + tx];
		Xs[tx][ty] = in[x + ty*m + tx];
		__syncthreads();
		
		for(k=0; k<SZ; k++) {
			tmp = Ys[ty][k] - Xs[k][tx];
			s += tmp*tmp;
		}
		__syncthreads();
	}
	o = by*SZ*n + ty*n + bx*SZ + tx;
	out[o] = s; // not sqrtf(s)
}

__global__ void 
gpuPdist_col( float* out, float* in, int rows, int cols )
{
	int i, squareeucldist = 0;
    //int c = blockDim.x * blockIdx.x + threadIdx.x; // cols
    int r = blockDim.y * blockIdx.y + threadIdx.y; // rows

	if( r < rows  ){
		for ( i = 0; i < cols; i++ ) //row-major order
            squareeucldist  += ( in[i + cols*r] - in[i + cols*r] ) * ( in[i + cols*r] - in[i + cols*r] );
		out[r] = squareeucldist;
		squareeucldist = 0;
    }
}